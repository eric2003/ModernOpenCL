#include "Info.h"
#include <cstdio>

void ShowCudaGpuInfo()
{
    int num_gpus = 0;    
    hipGetDeviceCount( &num_gpus );
    
    std::printf("CUDA : hipGetDeviceCount : number of CUDA devices:\t%d\n", num_gpus);
}
